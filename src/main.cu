#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <simple_clock.hpp>
#include "common.cuh"
#include "simulation.cuh"
#include "stb_image.h"
#include "stb_image_write.h"

#include <disp/window.hpp>
#include <drawables/buffer_drawable.hpp>
#include <iostream>
#include <memory>
#include <thread>

int main(int argc, char** argv) {
  if (argc != 2) {
    std::cout << "Wrong number of arguments\n" << std::endl;
    std::cout << "Please provide path to png file containing simulation setup to run" << std::endl;
    return 0;
  }
  std::string sim_output = "sim_output.png";
  std::string arg_str = argv[1];
  checkCudaErrors(hipSetDevice(0));

  std::cout << "Starting simulation\n" << std::endl;

  Simulation sim(arg_str);

  // setup drawable buffors
  auto sim_pixel_dims = sim.simulation_pixel_size();
  auto pix_art = std::make_shared<buffor_drawable<uint32_t>>(
      sim_pixel_dims.x, sim_pixel_dims.y, vec2<unsigned>{1u, 1u});
  disp::Window main_window("First window tests", vec2<int>{1280, 720});
  main_window.add_drawable(pix_art);

  if (main_window.initialise()) {
    return -1;
  }

  simple_clock sim_clock;
  using std::chrono::milliseconds;

  for (int iter = 0; iter < 10000; iter++) {
    main_window.update();
    if (main_window.should_close()) {
      std::cout << "window should close" << std::endl;
      sim.save(sim_output);
      return -1;
    }
    main_window.clear(0xFFU);
    long long dt = sim_clock.restart<milliseconds>().count();
    // SIMULATION RELATED STUFF
    sim.step(dt);
    {
      std::lock_guard<std::mutex> lock(pix_art->get_mtx());
      sim.put_pixel_data(*pix_art->get());
    }

    // SIMULATION RELATED STUFF END
    main_window.draw();
    std::cout << "dt: " << dt << std::endl;
  }
  std::cout << "sim done" << std::endl;
  sim.save(sim_output);

  while (!main_window.should_close()) {
    std::this_thread::sleep_for(std::chrono::milliseconds(20));
    main_window.update();
    std::this_thread::yield();
  }

  return 0;
}
