#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <simple_clock.hpp>
#include "common.cuh"
#include "simulation.cuh"
#include "stb_image.h"
#include "stb_image_write.h"

#include <disp/window.hpp>
#include <drawables/buffer_drawable.hpp>
#include <iostream>
#include <memory>
#include <thread>

int main(int argc, char** argv) {
  if (argc != 2) {
    std::cout << "Wrong number of arguments\n" << std::endl;
    return 0;
  }
  std::string sim_output = "sim_output.png";
  std::string arg_str = argv[1];

  std::cout << "Starting simulation\n" << std::endl;

  checkCudaErrors(hipSetDevice(0));

  Simulation sim(arg_str);

  sim.save(sim_output);
  auto sim_pixel_dims = sim.simulation_pixel_size();
  auto pix_art = std::make_shared<buffor_drawable<uint32_t>>(
      sim_pixel_dims.x, sim_pixel_dims.y, vec2<unsigned>{2u, 2u});
  disp::Window main_window("First window tests", vec2<int>{1280, 720});
  main_window.add_drawable(pix_art);

#ifdef DEBUG_DRAW_VISITED_PX
  auto debug_px = std::make_shared<buffor_drawable<uint32_t>>(sim_pixel_dims.x, sim_pixel_dims.y);
  main_window.add_drawable(debug_px);
#endif

  if (main_window.initialise()) {
    return -1;
  }

  simple_clock sim_clock;
  using std::chrono::milliseconds;

  for (int iter = 0; iter < 10000; iter++) {
    main_window.update();
    if (main_window.should_close()) {
      std::cout << "window should close" << std::endl;
      return -1;
    }
    main_window.clear(0xFFU);
    long long dt = sim_clock.restart<milliseconds>().count();
    // SIMULATION RELATED STUFF
    sim.step(dt);
    {
      std::lock_guard<std::mutex> lock(pix_art->get_mtx());
      sim.put_pixel_data(*pix_art->get());
    }
#ifdef DEBUG_DRAW_VISITED_PX
    {
      std::lock_guard<std::mutex> lock(debug_px->get_mtx());
      sim.put_visited_pixel_data(*debug_px->get());
    }
#endif
    // SIMULATION RELATED STUFF END
    main_window.draw();
    std::cout << "dt: " << dt << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(30));
  }
  std::cout << "donna" << std::endl;

  while (!main_window.should_close()) {
    std::this_thread::sleep_for(std::chrono::milliseconds(20));
    main_window.update();
    std::this_thread::yield();
  }

  // kernel<<<1,1>>>();
  return 0;
}
