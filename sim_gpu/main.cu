#include "gpu_sim.cuh"
#include "cuda_sim.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <iterator>
#include <memory.h>
#include <memory>
#include <vector>
#include <time.h>


typedef struct timespec app_timer_t;
#define timer(t_ptr) clock_gettime(CLOCK_MONOTONIC, t_ptr)
void elapsed_time(app_timer_t start, app_timer_t stop)
{
	double etime;
	etime = 1e+3 * (stop.tv_sec - start.tv_sec) +
		1e-6 * (stop.tv_nsec - start.tv_nsec);
	printf("GPU (total!) time = %.3f ms\n",
		etime);
}

__host__
static void sim_sand_kernel(GPUsim &sim){
    sim.simStep(1);
    //if(i<sim.getParticleNum()){
      //  sim.simStep(i);
     //   particles_p_[i].calculateNextLocation(time_step_, density_, viscosity_);
    //}
}

Sand *particle_vec_p;

int main(){
   // Sand particle_vec_p[sim_size];
   //     hipEvent_t start, stop;


    //float time_step = (float)MAX_TIME/SIM_STEPS;
    float n = 0.0000107; //dynamic viscosity of liquid  in Pa*s
    float p = 1;// Density of the liquid in kg/m3
    GPUsim testSim(n,p,
    MAX_TIME,SIM_STEPS,SIM_SIZE);
    //checkCudaErrors(hipSetDevice(0));
    std::cout<<"before:" << std::endl;
    testSim[1].printPosition();
    sim_sand_kernel(testSim);
    testSim.collect();
    std::cout<<"after:" << std::endl;
    testSim[1].printPosition();

    //checkCudaErrors(hipMalloc((void**)&particle_vec, (particle_vec.size()-1)*sizeof(Sand)));
    //checkCudaErrors(hipMemcpy(particle_vec_p, particle_vec, (particle_vec.size()-1)*sizeof(Sand), hipMemcpyHostToDevice));


   // elapsed_time(start,stop);
    return 0;
};

