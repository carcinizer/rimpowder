
#include "cuda_sim.cuh"
#include "gpu_sim.cuh"

#include <cstdint>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>



GPUsim::GPUsim(float viscosity, float density, float max_time,
 uint32_t max_it, uint32_t particle_num):
viscosity_(viscosity), density_(density), max_time_(max_time),
max_it_(max_it), particle_num_(particle_num) {

device_ = 0;
checkCudaErrors(hipSetDevice(device_));

particles_ = (Sand*)std::malloc(sizeof(Sand)*particle_num);

for(int i=0; i< particle_num_;i++){
//sand particle location should be changed
//but for simple test it should not matter
    particles_[i] = Sand(0,0);
}

checkCudaErrors(hipMalloc((void**)&particles_, 2*particle_num_* sizeof( Sand)));

checkCudaErrors(hipMemcpy(particles_p_, particles_, 
2*particle_num_* sizeof( Sand), hipMemcpyHostToDevice));

}

__device__  void GPUsim::sim_kernel(){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<particle_num_){
       particles_[i].calculateNextLocation(time_step_, density_, viscosity_);
    }
}

__host__  __device__ void GPUsim::simStep(int i){
    hipEvent_t start, stop;

    //particles_p_[i].calculateNextLocation(time_step_,
     //density_, viscosity_);

    //checkCudaErrors(hipEventCreate(&start));
	//checkCudaErrors(hipEventCreate(&stop));
	//checkCudaErrors(hipEventRecord(start, 0));

    sim_kernel<<<(particle_num_+K-1)/K, K>>>();    
}

uint32_t GPUsim::getParticleNum(){
    return particle_num_;
}



